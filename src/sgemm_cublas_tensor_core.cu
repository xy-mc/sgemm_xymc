#include "../include/sgemm_common.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

// 全局 handle，只创建一次
static hipblasHandle_t handle_tc = nullptr;

void sgemm_cublas_tensorcore(float* C, const float* A, const float* B, const MatrixDims& dims) {
    if (handle_tc == nullptr) {
        hipblasCreate(&handle_tc);
        // 启用TF32
        hipblasSetMathMode(handle_tc, HIPBLAS_TF32_TENSOR_OP_MATH);
    }

    int m = dims.M, n = dims.N, k = dims.K;
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // cuBLAS使用列主序，所以需要转置矩阵
    hipblasSgemm(
        handle_tc,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        n, m, k,
        &alpha,
        B, n,
        A, k,
        &beta,
        C, n
    );
}

// 清理函数，在程序结束时调用
void cleanup_cublas_tensorcore() {
    if (handle_tc != nullptr) {
        hipblasDestroy(handle_tc);
        handle_tc = nullptr;
    }
}