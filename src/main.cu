#include "../include/sgemm_common.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <iomanip>

// 声明不同版本的SGEMM实现
void sgemm_v0_global_memory(float* C, const float* A, const float* B, const MatrixDims& dims);
void sgemm_v1_shared_memory(float* C, const float* A, const float* B, const MatrixDims& dims);
void sgemm_v2_tiling(float* C, const float* A, const float* B, const MatrixDims& dims);
void sgemm_v3_vectorized(float* C, const float* A, const float* B, const MatrixDims& dims);
void sgemm_v4_register(float* C, const float* A, const float* B, const MatrixDims& dims);
void sgemm_v5_transpose(float* C, const float* A, const float* B, const MatrixDims& dims);
void sgemm_v6_double_buffer(float* C, const float* A, const float* B, const MatrixDims& dims);
void sgemm_v7_bank_conflict(float* C, const float* A, const float* B, const MatrixDims& dims);
void sgemm_cublas(float* C, const float* A, const float* B, const MatrixDims& dims);


// 运行cuBLAS并保存结果
void run_cublas_and_save_result(MatrixData& data) {
    // 创建临时数组来存储cuBLAS结果
    float* cublas_result;
    hipMalloc(&cublas_result, data.size_C);
    
    // 运行cuBLAS
    sgemm_cublas(cublas_result, data.d_A, data.d_B, data.dims);
    
    // 将结果复制回主机
    hipMemcpy(data.h_C, cublas_result, data.size_C, hipMemcpyDeviceToHost);
    
    // 清理临时数组
    hipFree(cublas_result);
}

// 将矩阵结果写入文件
void write_matrix_to_file(const std::string& filename, const float* matrix, int M, int N) {
    std::ofstream outfile(filename);
    if (!outfile.is_open()) {
        std::cerr << "Error: Could not open file " << filename << std::endl;
        return;
    }

    outfile << std::fixed << std::setprecision(6);
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            outfile << matrix[i * N + j] << " ";
        }
        outfile << "\n";
    }
    outfile.close();
}

int main() {
    // 设置矩阵维度
    std::vector<MatrixDims> test_cases = {
        {1024, 1024, 1024},
        {2048, 2048, 2048},
        {4096, 4096, 4096}
        // {8192, 8192, 8192},
        // {16384, 16384, 16384}
    };

    // 测试每个维度
    for (const auto& dims : test_cases) {
        std::cout << "\nTesting matrix dimensions: " 
                  << dims.M << "x" << dims.K << " * " 
                  << dims.K << "x" << dims.N << "\n"
                  << "========================================\n";

        // 创建并初始化矩阵数据
        MatrixData data(dims);
        data.initialize();
        data.copyToDevice();

        // 运行所有版本的SGEMM
        std::vector<PerformanceResult> results;
        
        // results.push_back(runPerformanceTest(sgemm_v0_global_memory, data, 5, "Global Memory"));
        // results.push_back(runPerformanceTest(sgemm_v1_shared_memory, data, 5, "Shared Memory"));
        // results.push_back(runPerformanceTest(sgemm_v2_tiling, data, 5, "Tiling"));
        results.push_back(runPerformanceTest(sgemm_v3_vectorized, data, 5, "Vectorized"));
        results.push_back(runPerformanceTest(sgemm_v4_register, data, 5, "Register"));
        results.push_back(runPerformanceTest(sgemm_v5_transpose, data, 5, "Transpose"));
        results.push_back(runPerformanceTest(sgemm_v6_double_buffer, data, 5, "Double Buffer"));
        results.push_back(runPerformanceTest(sgemm_v7_bank_conflict, data, 5, "Bank Conflict"));
        results.push_back(runPerformanceTest(sgemm_cublas, data, 5, "cuBLAS"));

        // 打印所有结果
        for (const auto& result : results) {
            printPerformanceResult(result);
        }

        // 进行误差测试
        std::cout << "\nError Analysis (compared with cuBLAS):\n"
                  << "========================================\n";
        
        std::vector<ErrorResult> error_results;
        // error_results.push_back(runErrorTest(sgemm_v0_global_memory, data, "Global Memory"));
        // error_results.push_back(runErrorTest(sgemm_v1_shared_memory, data, "Shared Memory"));
        // error_results.push_back(runErrorTest(sgemm_v2_tiling, data, "Tiling"));
        error_results.push_back(runErrorTest(sgemm_v3_vectorized, data, "Vectorized"));
        error_results.push_back(runErrorTest(sgemm_v4_register, data, "Register"));
        error_results.push_back(runErrorTest(sgemm_v5_transpose, data, "Transpose"));
        error_results.push_back(runErrorTest(sgemm_v6_double_buffer, data, "Double Buffer"));
        error_results.push_back(runErrorTest(sgemm_v7_bank_conflict, data, "Bank Conflict"));
        for (const auto& result : error_results) {
            printErrorResult(result);
        }

        // // 输出最后一个计算结果到文件
        // data.copyToHost();
        // std::string last_result_file = "last_result_" + std::to_string(dims.M) + "x" + std::to_string(dims.N) + ".txt";
        // write_matrix_to_file(last_result_file, data.h_C, dims.M, dims.N);
        // std::cout << "\nLast computation results written to: " << last_result_file << std::endl;

        // // 运行cuBLAS并输出结果到文件
        // run_cublas_and_save_result(data);
        // std::string cublas_result_file = "cublas_result_" + std::to_string(dims.M) + "x" + std::to_string(dims.N) + ".txt";
        // write_matrix_to_file(cublas_result_file, data.h_C, dims.M, dims.N);
        // std::cout << "cuBLAS results written to: " << cublas_result_file << std::endl;
    }

    return 0;
} 