#include "hip/hip_runtime.h"
#include "../include/sgemm_common.h"

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FLOAT4(ptr) (reinterpret_cast<float4*>(&(ptr))[0])
#define CONST_FLOAT4(ptr) (reinterpret_cast<const float4*>(&(ptr))[0])

#define BLOCK_SIZE 16
#define TILE_SIZE 4

// constexpr int STEP = BLOCK_SIZE * TILE_SIZE;

__global__ void sgemm_vectorized_kernel(
    float* C,
    const float* A,
    const float* B,
    int M, int N, int K) {
    
    const int BM = 128;
    const int BN = 128;
    const int BK = 8;

    const int TM = BM / BLOCK_SIZE;
    const int TN = BN / BLOCK_SIZE;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    __shared__ float A_shared[BM][BK];
    __shared__ float B_shared[BK][BN];
    
    float sum[TM][TN] = {0.0f};
    
    const float *A_start = A + by * BM * K;
    const float *B_start = B + bx * BN;
    
    const int sy_k = (ty * BLOCK_SIZE + tx) * 4 / BK;
    const int sx_k = (ty * BLOCK_SIZE + tx) * 4 % BK;

    const int sy_n = (ty * BLOCK_SIZE + tx) * 4 / BN;
    const int sx_n = (ty * BLOCK_SIZE + tx) * 4 % BN;

    #pragma unroll
    for (int s = 0; s < K; s += BK) {

        FLOAT4(A_shared[sy_k][sx_k])
                = CONST_FLOAT4(A_start[OFFSET(sy_k, sx_k + s, K)]);

        FLOAT4(B_shared[sy_n][sx_n])
                = CONST_FLOAT4(B_start[OFFSET(sy_n + s, sx_n, N)]);
            
        __syncthreads();

        #pragma unroll
        for (int k = 0; k < BK; k++) {
            for (int i = 0; i < TM; i++) {
                for (int j = 0; j < TN; j++) {

                    // sum[i][j] += A_shared[ty + i * BLOCK_SIZE][k] * B_shared[k][tx + j * BLOCK_SIZE];
                    sum[i][j] += A_shared[ty * TM + i][k] * B_shared[k][tx * TN + j];
                    
                }
            }
        }
        __syncthreads();
    }

    float *C_start = C + by * BM * N + bx * BN;

    #pragma unroll
    for (int i = 0; i < TM; i++) {

        for (int j = 0; j < TN; j += 4) {

            FLOAT4(C_start[OFFSET(ty * TM + i, tx * TN + j, N)])
                    = FLOAT4(sum[i][j]);
            // C_start[OFFSET(ty + i * BLOCK_SIZE, tx + j * BLOCK_SIZE, N)]
            //         = sum[i][j];
        }
    }
    
}

void sgemm_v3_vectorized(float* C, const float* A, const float* B, const MatrixDims& dims) {
    
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

    dim3 gridDim(
        (dims.N + blockDim.x - 1) / blockDim.x / 8,
        (dims.M + blockDim.y - 1) / blockDim.y / 8
    );
    
    sgemm_vectorized_kernel<<<gridDim, blockDim>>>(C, A, B, dims.M, dims.N, dims.K);
    
    hipError_t error = hipGetLastError(); 
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
}