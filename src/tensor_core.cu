#include "hip/hip_runtime.h"
#include "../include/sgemm_common.h"
#include <mma.h>

using namespace nvcuda;

#define WARP_SIZE 32
#define DEVICE_INLINE __device__ inline
#define HOST_DEVICE_INLINE __device__ __host__ inline
#define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162 *>(&(value))[0])
#define LDST32BITS(value) (reinterpret_cast<half2 *>(&(value))[0])
#define LDST64BITS(value) (reinterpret_cast<float2 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n)                                                 \
  asm volatile("cp.async.wait_group %0;\n" ::"n"(n))
// ca(cache all, L1 + L2): support 4, 8, 16 bytes, cg(cache global, L2): only
// support 16 bytes.
#define CP_ASYNC_CA(dst, src, bytes)                                           \
  asm volatile(                                                                \
      "cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst),       \
      "l"(src), "n"(bytes))
#define CP_ASYNC_CG(dst, src, bytes)                                           \
  asm volatile(                                                                \
      "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst),       \
      "l"(src), "n"(bytes))
// Support A and B matrix with row-major inorder to compare with the kernels
// using CUDA Cores in sgemm.cu and sgemm_async.cu. also need flag when
// compiling.

HOST_DEVICE_INLINE
int div_ceil(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

__global__ void f32x4_tf32x4_kernel(float *x, float *y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
  if (idx < N) {
    float4 reg_x = FLOAT4(x[idx]);
    float4 reg_y;
    reg_y.x = wmma::__float_to_tf32(reg_x.x);
    reg_y.y = wmma::__float_to_tf32(reg_x.y);
    reg_y.z = wmma::__float_to_tf32(reg_x.z);
    reg_y.w = wmma::__float_to_tf32(reg_x.w);
    FLOAT4(y[idx]) = reg_y;
  }
}
// stage2/3/4 (stage2=double buffers+copy async)
// 1. When using shared memory exceeds 48 KB, dynamic shared memory needs to be
// used, i.e., declare a block of dynamic shared memory with extern shared half
// smem[];. When calling the kernel, the size of the dynamic shared memory needs
// to be specified, and smem addressing should be used in a one-dimensional
// array manner.
// 2. Improve L2 Cache locality (Thread Block Swizzle):
// https://zhuanlan.zhihu.com/p/555339335
// 3. __launch_bounds__: avoid error 'too many resources required for launch'
// reference: https://blog.csdn.net/feng__shuai/article/details/124395023
template <const int WMMA_M = 16, const int WMMA_N = 16, const int WMMA_K = 8,
          const int WMMA_TILE_M = 4, const int WMMA_TILE_N = 2,
          const int WARP_TILE_M = 2, const int WARP_TILE_N = 4,
          const int A_PAD = 0, const int B_PAD = 0, const int K_STAGE = 2,
          const bool BLOCK_SWIZZLE = false>
__global__ void
sgemm_wmma_m16n16k8_mma4x2_warp2x4_stages_kernel(const float *A, const float *B, float *C,
                                                 int M, int N, int K) {
  // 256 threads(8 warps) per block.
  // const int bx = blockIdx.x;
  // BLOCK_SWIZZLE 0/1 控制是否使用 block swizzle
  const int bx = ((int)BLOCK_SWIZZLE) * blockIdx.z * gridDim.x + blockIdx.x;
  const int by = blockIdx.y;
  const int NUM_K_TILES = div_ceil(K, WMMA_K);
  constexpr int BM = WMMA_M * WMMA_TILE_M * WARP_TILE_M; // 16x4*2=128
  constexpr int BN = WMMA_N * WMMA_TILE_N * WARP_TILE_N; // 16x2*4=128
  constexpr int BK = WMMA_K;                             // 8
  __shared__ float s_a[K_STAGE][BM][BK + A_PAD], s_b[K_STAGE][BK][BN + B_PAD];

  // 要保证相同的warp下thread执行相同的指令
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  const int warp_id = tid / WARP_SIZE; // 0~7 warp_id within block
  const int warp_m = warp_id / 2;      // 0,1,2,3
  const int warp_n = warp_id % 2;      // 0,1

  // 先计算shared memory中的索引
  // tid和需要加载的smem s_a[BM][BK] 之间的索引关系 BM=128 BK=8 按行读取 A行主序
  // 对于s_a每行8个数据，每个线程读取4个，需要2个线程；总共128行，需要128x2刚好256线程
  int load_smem_a_m = tid / 2;                // row 0~127
  int load_smem_a_k = (tid % 2 == 0) ? 0 : 4; // col 0,4
  // tid和需要加载的smem s_b[BK][BN] 之间的索引关系 BK=8 BN=128 按行读取 B行主序
  // 对于s_b每行128个数据，每个线程读4个数据，需要32个线程；总共8行，需要32x8=256个线程
  int load_smem_b_k = tid / 32;       // row 0~7
  int load_smem_b_n = (tid % 32) * 4; // col 0,4,...,124,...
  // 再计算全局内存中的索引
  // 要加载到s_a中的元素对应到A全局内存中的行数
  // 每个block负责出C中大小为BM*BN的块
  int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
  int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c

  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float>
      C_frag[WARP_TILE_M][WARP_TILE_N];

#pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      wmma::fill_fragment(C_frag[i][j], 0.0);
    }
  }

#pragma unroll
  for (int k = 0; k < (K_STAGE - 1); ++k) {         // 0, 1
    int load_gmem_a_k = k * WMMA_K + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * WMMA_K + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n;

    uint32_t load_smem_a_ptr =
        __cvta_generic_to_shared(&s_a[k][load_smem_a_m][load_smem_a_k]);
    CP_ASYNC_CG(load_smem_a_ptr, &A[load_gmem_a_addr], 16);

    uint32_t load_smem_b_ptr =
        __cvta_generic_to_shared(&s_b[k][load_smem_b_k][load_smem_b_n]);
    CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);

    CP_ASYNC_COMMIT_GROUP();
  }

  CP_ASYNC_WAIT_GROUP(K_STAGE - 2); // s2->0, s3->1, s4->2
  __syncthreads();

#pragma unroll
  for (int k = (K_STAGE - 1); k < NUM_K_TILES; k++) {
    // s2/4 can use bitwise ops but s3 can not, so, we use mod
    // ops for all stages kernel. s2: (k + 1)&1, s4: (k + 1)&3
    // s3: (k + 1) % 3
    int smem_sel = (k + 1) % K_STAGE; // s3 k 2->0, k 3->1, k 4->2...
    int smem_sel_next = k % K_STAGE;  // s3 k 2->2, k 3->0, k 4->1...

    // k * WMMA_K, WMMA_K=16 -> (k << 4)
    int load_gmem_a_k = k * WMMA_K + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * WMMA_K + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n;

    // load stage 2, k start from 2
    uint32_t load_smem_a_ptr = __cvta_generic_to_shared(
        &s_a[smem_sel_next][load_smem_a_m][load_smem_a_k]);
    CP_ASYNC_CG(load_smem_a_ptr, &A[load_gmem_a_addr], 16);

    uint32_t load_smem_b_ptr = __cvta_generic_to_shared(
        &s_b[smem_sel_next][load_smem_b_k][load_smem_b_n]);
    CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);
    CP_ASYNC_COMMIT_GROUP();

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K,
                   wmma::precision::tf32, wmma::row_major>
        A_frag[WARP_TILE_M];
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K,
                   wmma::precision::tf32, wmma::row_major>
        B_frag[WARP_TILE_N];

// compute stage 0
#pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
      // load 2 tiles -> reg, smem a -> frags a, warp_m 0~3
      const int warp_smem_a_m = warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
      wmma::load_matrix_sync(A_frag[i], &s_a[smem_sel][warp_smem_a_m][0],
                             BK + A_PAD);
    }

#pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      // load 4 tiles -> reg, smem b -> frags b, warp_n 0~2
      const int warp_smem_b_n = warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
      wmma::load_matrix_sync(B_frag[j], &s_b[smem_sel][0][warp_smem_b_n],
                             BN + B_PAD);
    }

#pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        wmma::mma_sync(C_frag[i][j], A_frag[i], B_frag[j], C_frag[i][j]);
      }
    }

    CP_ASYNC_WAIT_GROUP(K_STAGE - 2);
    __syncthreads();
  }

  // make sure all memory issues ready.
  if ((K_STAGE - 2) > 0) {
    CP_ASYNC_WAIT_GROUP(0);
    __syncthreads();
  }
  // processing last (K_STAGE-1) k iters.
  {
#pragma unroll
    for (int k = 0; k < (K_STAGE - 1); k++) {
      const int stage_sel = ((NUM_K_TILES - (K_STAGE - 1) + k) % K_STAGE);
      wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K,
                     wmma::precision::tf32, wmma::row_major>
          A_frag[WARP_TILE_M];
      wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K,
                     wmma::precision::tf32, wmma::row_major>
          B_frag[WARP_TILE_N];

#pragma unroll
      for (int i = 0; i < WARP_TILE_M; ++i) {
        // load 2 tiles -> reg, smem a -> frags a, warp_m 0~3
        const int warp_smem_a_m = warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
        wmma::load_matrix_sync(A_frag[i], &s_a[stage_sel][warp_smem_a_m][0],
                               BK + A_PAD);
      }

#pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        // load 4 tiles -> reg, smem b -> frags b, warp_n 0~2
        const int warp_smem_b_n = warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
        wmma::load_matrix_sync(B_frag[j], &s_b[stage_sel][0][warp_smem_b_n],
                               BN + B_PAD);
      }

#pragma unroll
      for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
        for (int j = 0; j < WARP_TILE_N; ++j) {
          wmma::mma_sync(C_frag[i][j], A_frag[i], B_frag[j], C_frag[i][j]);
        }
      }
    }
  }

// finally, store back to C matrix.
#pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      const int store_gmem_a_m =
          by * BM + warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
      const int store_gmem_a_n =
          bx * BN + warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
      wmma::store_matrix_sync(C + store_gmem_a_m * N + store_gmem_a_n,
                              C_frag[i][j], N, wmma::mem_row_major);
    }
  }
}

#define BLOCK_SIZE 16

void sgemm_tensor_example(float* C, const float* A, const float* B, const MatrixDims& dims) {
    
    dim3 blockDim(BLOCK_SIZE * BLOCK_SIZE);

    dim3 gridDim(
        (dims.N + BLOCK_SIZE - 1) / BLOCK_SIZE / 8,
        (dims.M + BLOCK_SIZE - 1) / BLOCK_SIZE / 8
    );
    
    // hipFuncSetAttribute(reinterpret_cast<const void*>(
    //     sgemm_tensor_core_vectorized_kernel),
    //     hipFuncAttributeMaxDynamicSharedMemorySize,
    //     98304 // 64KB
    // );
    
    sgemm_wmma_m16n16k8_mma4x2_warp2x4_stages_kernel<16, 16, 8, 4, 2, 2, 4, 0, 0, 2, 0><<<gridDim, blockDim>>>(A, B, C, dims.M, dims.N, dims.K);
        
    hipError_t error = hipGetLastError(); 
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
}