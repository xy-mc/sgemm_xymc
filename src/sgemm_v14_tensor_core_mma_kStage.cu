#include "hip/hip_runtime.h"
#include "../include/sgemm_common.h"
#include "../include/PTX.h"
#include <mma.h>
#include <cstdio>
using namespace nvcuda;

#define BLOCK_SIZE 16

#define padding_A 0
#define padding_B 0

constexpr int Warp_M = 4;
constexpr int Warp_N = 4;

static __device__ 
uint32_t swizzle_A(uint32_t y, uint32_t x) {
    x >>= 2;
    return (((y & 7) >> 2) ^ x) << 2;
}

static __device__ 
uint32_t swizzle_B(uint32_t y, uint32_t x) {
    x >>= 3;
    return ((y & 3) ^ x) << 3;
}

template<const int Stage_num>
__global__ void sgemm_tensor_core_mma_kStage_kernel(
    float* C,
    const float* A,
    const float* B,
    int M, int N, int K) {

    constexpr int BM = 128;
    constexpr int BN = 128;
    constexpr int BK = 8; 

    const int bx = blockIdx.x;
    const int by = blockIdx.y;

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    const int warp_id = (ty * BLOCK_SIZE + tx) / 32;
    const int lane_id = (ty * BLOCK_SIZE + tx) % 32; 

    const int warp_m = warp_id / 4; // 0,1
    const int warp_n = warp_id % 4; // 0,1,2,3

    const int NUM_K_TILES = (K + BK - 1) / BK;

    __shared__ float A_shared[Stage_num][BM][BK + padding_A];
    __shared__ float B_shared[Stage_num][BK][BN + padding_B];
    
    uint32_t RA[Warp_M][4];

    uint32_t RB[Warp_N][2];

    float RC[Warp_M][Warp_N][4];

    #pragma unroll
    for (int i = 0; i < Warp_M; i++) {
        for (int j = 0; j < Warp_N; j++) {
            for (int k = 0; k < 4; k++) {
                RC[i][j][k] = 0.0f;
            }
        }
    }
    
    const float *A_start = A + by * BM * K;
    const float *B_start = B + bx * BN;
    
    const int sy_k = (ty * BLOCK_SIZE + tx) * 4 / BK;
    const int sx_k = (ty * BLOCK_SIZE + tx) * 4 % BK;

    const int sy_n = (ty * BLOCK_SIZE + tx) * 4 / BN;
    const int sx_n = (ty * BLOCK_SIZE + tx) * 4 % BN;
    
    #pragma unroll
    for (int k = 0; k < Stage_num - 1; k++) {

        uint32_t smem_ptr_A = __cvta_generic_to_shared(&A_shared[k][sy_k][swizzle_A(sy_k, sx_k)]);

        uint32_t smem_ptr_B = __cvta_generic_to_shared(&B_shared[k][sy_n][swizzle_B(sy_n, sx_n) + (sx_n % 8)]);

        CP_ASYNC_CG(smem_ptr_A, &A_start[OFFSET(sy_k, sx_k + k * BK, K)], 16);

        CP_ASYNC_CG(smem_ptr_B, &B_start[OFFSET(sy_n + k * BK, sx_n, N)], 16);

        CP_ASYNC_COMMIT_GROUP();
    }

    CP_ASYNC_WAIT_GROUP(Stage_num - 2);
    __syncthreads();

    // int reg_store_idx = 0;
    // int reg_load_idx = 1;
    int smem_reg = 0;

    // {
    //     #pragma unroll
    //     for (int i = 0; i < Warp_M; i++) {
            
    //         int smem_regA_addr_y = lane_id % 16;
    //         int smem_regA_addr_x = lane_id / 16 * 4;

    //         uint32_t smem_reg_A = 
    //             __cvta_generic_to_shared(&A_shared[smem_reg][16 * (i + warp_m * Warp_M) + smem_regA_addr_y]
    //                                                 [swizzle_A(smem_regA_addr_y, smem_regA_addr_x)]);
            
    //         LDMATRIX_X4(RA[reg_store_idx][i][0], RA[reg_store_idx][i][1], 
    //                     RA[reg_store_idx][i][2], RA[reg_store_idx][i][3], smem_reg_A);

    //     }

    //     #pragma unroll
    //     for (int i = 0; i < Warp_N; i++) {

    //         int smem_regB_addr_y = lane_id % 4;
    //         int smem_regB_addr_x = 8 * (i + warp_n * Warp_N);

    //         RB[reg_store_idx][i][0] = (uint32_t&)B_shared[smem_reg][smem_regB_addr_y]
    //                                 [swizzle_B(smem_regB_addr_y, smem_regB_addr_x) + lane_id / 4];

    //         RB[reg_store_idx][i][1] = (uint32_t&)B_shared[smem_reg][smem_regB_addr_y + 4]
    //                                 [swizzle_B(smem_regB_addr_y, smem_regB_addr_x) + lane_id / 4];
    //     }
    // }

    #pragma unroll
    for (int k = Stage_num - 1; k < NUM_K_TILES; k++) {
        
        #pragma unroll
        for (int i = 0; i < Warp_M; i++) {
            
            int smem_regA_addr_y = lane_id % 16;
            int smem_regA_addr_x = lane_id / 16 * 4;

            uint32_t smem_reg_A = 
                __cvta_generic_to_shared(&A_shared[smem_reg][16 * (i + warp_m * Warp_M) + smem_regA_addr_y]
                                                    [swizzle_A(smem_regA_addr_y, smem_regA_addr_x)]);
            
            LDMATRIX_X4(RA[i][0], RA[i][1], RA[i][2], RA[i][3], smem_reg_A);

        }

        #pragma unroll
        for (int i = 0; i < Warp_N; i++) {

            int smem_regB_addr_y = lane_id % 4;
            int smem_regB_addr_x = 8 * (i + warp_n * Warp_N);

            RB[i][0] = (uint32_t&)B_shared[smem_reg][smem_regB_addr_y]
                                    [swizzle_B(smem_regB_addr_y, smem_regB_addr_x) + lane_id / 4];

            RB[i][1] = (uint32_t&)B_shared[smem_reg][smem_regB_addr_y + 4]
                                    [swizzle_B(smem_regB_addr_y, smem_regB_addr_x) + lane_id / 4];
        }

        #pragma unroll
        for (int i = 0; i < Warp_M; i++) {
            
            #pragma unroll
            for (int j = 0; j < Warp_N; j++) {

                SMMA1688(RC[i][j][0], RC[i][j][1], RC[i][j][2], RC[i][j][3], 
                    RA[i][0], RA[i][1], RA[i][2], RA[i][3], 
                    RB[j][0], RB[j][1], 
                    RC[i][j][0], RC[i][j][1], RC[i][j][2], RC[i][j][3]);

            }

        }

        int global_smem = k % Stage_num;

        uint32_t smem_ptr_A = 
                __cvta_generic_to_shared(&A_shared[global_smem][sy_k][swizzle_A(sy_k, sx_k)]);

        uint32_t smem_ptr_B = 
                __cvta_generic_to_shared(&B_shared[global_smem][sy_n][swizzle_B(sy_n, sx_n) + (sx_n % 8)]);

        CP_ASYNC_CG(smem_ptr_A, &A_start[OFFSET(sy_k, sx_k + k * BK, K)], 16);

        CP_ASYNC_CG(smem_ptr_B, &B_start[OFFSET(sy_n + k * BK, sx_n, N)], 16);

        CP_ASYNC_COMMIT_GROUP();

        // #pragma unroll
        // for (int i = 0; i < Warp_M; i++) {
            
        //     int smem_regA_addr_y = lane_id % 16;
        //     int smem_regA_addr_x = lane_id / 16 * 4;

        //     uint32_t smem_reg_A = 
        //         __cvta_generic_to_shared(&A_shared[smem_reg][16 * (i + warp_m * Warp_M) + smem_regA_addr_y]
        //                                             [swizzle_A(smem_regA_addr_y, smem_regA_addr_x)]);
            
        //     LDMATRIX_X4(RA[i][0], RA[i][1], RA[i][2], RA[i][3], smem_reg_A);

        // }

        // #pragma unroll
        // for (int i = 0; i < Warp_N; i++) {

        //     int smem_regB_addr_y = lane_id % 4;
        //     int smem_regB_addr_x = 8 * (i + warp_n * Warp_N);

        //     RB[i][0] = (uint32_t&)B_shared[smem_reg][smem_regB_addr_y]
        //                             [swizzle_B(smem_regB_addr_y, smem_regB_addr_x) + lane_id / 4];

        //     RB[i][1] = (uint32_t&)B_shared[smem_reg][smem_regB_addr_y + 4]
        //                             [swizzle_B(smem_regB_addr_y, smem_regB_addr_x) + lane_id / 4];
        // }

        // #pragma unroll
        // for (int i = 0; i < Warp_M; i++) {
            
        //     #pragma unroll
        //     for (int j = 0; j < Warp_N; j++) {

        //         SMMA1688(RC[i][j][0], RC[i][j][1], RC[i][j][2], RC[i][j][3], 
        //             RA[i][0], RA[i][1], RA[i][2], RA[i][3], 
        //             RB[j][0], RB[j][1], 
        //             RC[i][j][0], RC[i][j][1], RC[i][j][2], RC[i][j][3]);

        //     }

        // }

        smem_reg = (smem_reg + 1) % Stage_num;

        CP_ASYNC_WAIT_GROUP(Stage_num - 2);
        __syncthreads();
    }

    if constexpr ((Stage_num - 2) > 0) {
        CP_ASYNC_WAIT_GROUP(0);
        __syncthreads();
    }

    #pragma unroll
    for (int k = 0; k < Stage_num - 1; k++) {

        // reg_store_idx ^= 1;
        // reg_load_idx ^= 1;
        

        #pragma unroll
        for (int i = 0; i < Warp_M; i++) {
            
            int smem_regA_addr_y = lane_id % 16;
            int smem_regA_addr_x = lane_id / 16 * 4;

            uint32_t smem_reg_A = 
                __cvta_generic_to_shared(&A_shared[smem_reg][16 * (i + warp_m * Warp_M) + smem_regA_addr_y]
                                                    [swizzle_A(smem_regA_addr_y, smem_regA_addr_x)]);
            
            LDMATRIX_X4(RA[i][0], RA[i][1], RA[i][2], RA[i][3], smem_reg_A);

            #pragma unroll
            for (int j = 0; j < Warp_N; j++) {

                int smem_regB_addr_y = lane_id % 4;
                int smem_regB_addr_x = 8 * (i + warp_n * Warp_N);

                RB[i][0] = (uint32_t&)B_shared[smem_reg][smem_regB_addr_y]
                                        [swizzle_B(smem_regB_addr_y, smem_regB_addr_x) + lane_id / 4];

                RB[i][1] = (uint32_t&)B_shared[smem_reg][smem_regB_addr_y + 4]
                                        [swizzle_B(smem_regB_addr_y, smem_regB_addr_x) + lane_id / 4];
                                    
            }

        }

        #pragma unroll
        for (int i = 0; i < Warp_N; i++) {

            int smem_regB_addr_y = lane_id % 4;
            int smem_regB_addr_x = 8 * (i + warp_n * Warp_N);

            RB[i][0] = (uint32_t&)B_shared[smem_reg][smem_regB_addr_y]
                                    [swizzle_B(smem_regB_addr_y, smem_regB_addr_x) + lane_id / 4];

            RB[i][1] = (uint32_t&)B_shared[smem_reg][smem_regB_addr_y + 4]
                                    [swizzle_B(smem_regB_addr_y, smem_regB_addr_x) + lane_id / 4];
                                    
        }

        #pragma unroll
        for (int i = 0; i < Warp_M; i++) {
            
            #pragma unroll
            for (int j = 0; j < Warp_N; j++) {

                SMMA1688(RC[i][j][0], RC[i][j][1], RC[i][j][2], RC[i][j][3], 
                    RA[i][0], RA[i][1], RA[i][2], RA[i][3], 
                    RB[j][0], RB[j][1], 
                    RC[i][j][0], RC[i][j][1], RC[i][j][2], RC[i][j][3]);

            }
        }

        smem_reg = (smem_reg + 1) % Stage_num;
    }

    float *C_start = C + by * BM * N + bx * BN + warp_m * Warp_M * 16 * N + warp_n * Warp_N * 8;

    #pragma unroll
    for (int i = 0; i < Warp_M; i++) {

        #pragma unroll
        for (int j = 0; j < Warp_N; j++) {
            
            LDST64BITS(C_start[OFFSET(16 * i + lane_id / 4, 8 * j + (lane_id % 4) * 2, N)]) 
                        = LDST64BITS(RC[i][j][0]);

            LDST64BITS(C_start[OFFSET(16 * i + lane_id / 4 + 8, 8 * j + (lane_id % 4) * 2, N)]) 
                        = LDST64BITS(RC[i][j][2]);

        }

    }
    
    
}

void sgemm_v14_tensor_core_mma_kStage(float* C, const float* A, const float* B, const MatrixDims& dims) {
    
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

    dim3 gridDim(
        (dims.N + blockDim.x - 1) / blockDim.x / 8,
        (dims.M + blockDim.y - 1) / blockDim.y / 8
    );
    
    sgemm_tensor_core_mma_kStage_kernel<4><<<gridDim, blockDim>>>(C, A, B, dims.M, dims.N, dims.K);
    
    hipError_t error = hipGetLastError(); 
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
}